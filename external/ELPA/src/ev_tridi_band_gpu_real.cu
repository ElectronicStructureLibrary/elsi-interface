//    This file is part of ELPA.
//
//    The ELPA library was originally created by the ELPA consortium,
//    consisting of the following organizations:
//
//    - Max Planck Computing and Data Facility (MPCDF), formerly known as
//      Rechenzentrum Garching der Max-Planck-Gesellschaft (RZG),
//    - Bergische Universität Wuppertal, Lehrstuhl für angewandte
//      Informatik,
//    - Technische Universität München, Lehrstuhl für Informatik mit
//      Schwerpunkt Wissenschaftliches Rechnen ,
//    - Fritz-Haber-Institut, Berlin, Abt. Theorie,
//    - Max-Plack-Institut für Mathematik in den Naturwissenschaften,
//      Leipzig, Abt. Komplexe Strukutren in Biologie und Kognition,
//      and
//    - IBM Deutschland GmbH
//
//    This particular source code file contains additions, changes and
//    enhancements authored by Intel Corporation which is not part of
//    the ELPA consortium.
//
//    More information can be found here:
//    http://elpa.mpcdf.mpg.de/
//
//    ELPA is free software: you can redistribute it and/or modify
//    it under the terms of the version 3 of the license of the
//    GNU Lesser General Public License as published by the Free
//    Software Foundation.
//
//    ELPA is distributed in the hope that it will be useful,
//    but WITHOUT ANY WARRANTY; without even the implied warranty of
//    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//    GNU Lesser General Public License for more details.
//
//    You should have received a copy of the GNU Lesser General Public License
//    along with ELPA.  If not, see <http://www.gnu.org/licenses/>
//
//    ELPA reflects a substantial effort on the part of the original
//    ELPA consortium, and we ask you to respect the spirit of the
//    license that we chose: i.e., please contribute any changes you
//    may have back to the original ELPA library distribution, and keep
//    any derivatives of ELPA under the same license that we chose for
//    the original distribution, the GNU Lesser General Public License.
//
//

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#if (CUDART_VERSION >= 9000)
template <typename T, unsigned int blk> __device__ void warp_shfl_reduce_real(volatile T *s_block)
{
    unsigned int tid = threadIdx.x;

    T val;

    if (blk >= 64)
    {
        if (tid < 32)
        {
            s_block[tid] += s_block[tid + 32];
        }
    }

    val = s_block[tid];

    for (int i = 16; i >= 1; i /= 2)
    {
        val += __shfl_xor_sync(0xffffffff, val, i, 32);
    }

    s_block[tid] = val;
}
#endif

template <typename T, unsigned int blk> __device__ void warp_reduce_real(volatile T *s_block)
{
    unsigned int tid = threadIdx.x;

    if (blk >= 64)
    {
        if (tid < 32)
        {
            s_block[tid] += s_block[tid + 32];
        }
    }

    if (blk >= 32)
    {
        if (tid < 16)
        {
            s_block[tid] += s_block[tid + 16];
        }
    }

    if (blk >= 16)
    {
        if (tid < 8)
        {
            s_block[tid] += s_block[tid + 8];
        }
    }

    if (blk >= 8)
    {
        if (tid < 4)
        {
            s_block[tid] += s_block[tid + 4];
        }
    }

    if (blk >= 4)
    {
        if (tid < 2)
        {
            s_block[tid] += s_block[tid + 2];
        }
    }

    if (blk >= 2)
    {
        if (tid < 1)
        {
            s_block[tid] += s_block[tid + 1];
        }
    }
}

template <typename T, unsigned int blk> __device__ void reduce_real(T *s_block)
{
    unsigned int tid = threadIdx.x;

    if (blk >= 1024)
    {
        if (tid < 512)
        {
            s_block[tid] += s_block[tid + 512];
        }

        __syncthreads();
    }

    if (blk >= 512)
    {
        if (tid < 256)
        {
            s_block[tid] += s_block[tid + 256];
        }

        __syncthreads();
    }

    if (blk >= 256)
    {
        if (tid < 128)
        {
            s_block[tid] += s_block[tid + 128];
        }

        __syncthreads();
    }

    if (blk >= 128)
    {
        if (tid < 64)
        {
            s_block[tid] += s_block[tid + 64];
        }

        __syncthreads();
    }

#if (CUDART_VERSION > 9000)
    if (blk >= 32)
    {
        if (tid < 32)
        {
            warp_shfl_reduce_real<T, blk>(s_block);
        }
    }
    else
    {
        if (tid < 32)
        {
            warp_reduce_real<T, blk>(s_block);
        }
    }
#else
    if (tid < 32)
    {
        warp_reduce_real<T, blk>(s_block);
    }
#endif
}

template <typename T, unsigned int blk>
__global__ void compute_hh_trafo_kernel_real(T * __restrict__ q, const T * __restrict__ hh, const T * __restrict__ hh_tau, const int nb, const int ldq, const int ncols)
{
    __shared__ T q_s[blk + 1];
    __shared__ T dotp_s[blk];

    T q_v2;

    int q_off, h_off, j;

    unsigned int tid = threadIdx.x;
    unsigned int bid = blockIdx.x;

    j = ncols;
    q_off = bid + (j + tid) * ldq;
    h_off = tid + j * nb;
    q_s[tid] = q[q_off];

    while (j >= 1)
    {
        if (tid == 0)
        {
            q_s[tid] = q[q_off];
        }

        q_v2 = q_s[tid];
        dotp_s[tid] = q_v2 * hh[h_off];

        __syncthreads();

        reduce_real<T, blk>(dotp_s);

        __syncthreads();

        q_v2 -= dotp_s[0] * hh_tau[j] * hh[h_off];
        q_s[tid + 1] = q_v2;

        if ((j == 1) || (tid == blockDim.x - 1))
        {
            q[q_off] = q_v2;
        }

        __syncthreads();

        q_off -= ldq;
        h_off -= nb;
        j -= 1;
    }
}

extern "C" void launch_compute_hh_trafo_c_kernel_real_double(double *q, const double *hh, const double *hh_tau, const int nev, const int nb, const int ldq, const int ncols)
{
    hipError_t err;

    switch (nb)
    {
    case 1024:
        compute_hh_trafo_kernel_real<double, 1024><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 512:
        compute_hh_trafo_kernel_real<double, 512><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 256:
        compute_hh_trafo_kernel_real<double, 256><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 128:
        compute_hh_trafo_kernel_real<double, 128><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 64:
        compute_hh_trafo_kernel_real<double, 64><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 32:
        compute_hh_trafo_kernel_real<double, 32><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 16:
        compute_hh_trafo_kernel_real<double, 16><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 8:
        compute_hh_trafo_kernel_real<double, 8><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 4:
        compute_hh_trafo_kernel_real<double, 4><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 2:
        compute_hh_trafo_kernel_real<double, 2><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 1:
        compute_hh_trafo_kernel_real<double, 1><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    }

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("\n compute_hh_trafo CUDA kernel failed: %s \n",hipGetErrorString(err));
    }
}

extern "C" void launch_compute_hh_trafo_c_kernel_real_single(float *q, const float *hh, const float *hh_tau, const int nev, const int nb, const int ldq, const int ncols)
{
    hipError_t err;

    switch (nb)
    {
    case 1024:
        compute_hh_trafo_kernel_real<float, 1024><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 512:
        compute_hh_trafo_kernel_real<float, 512><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 256:
        compute_hh_trafo_kernel_real<float, 256><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 128:
        compute_hh_trafo_kernel_real<float, 128><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 64:
        compute_hh_trafo_kernel_real<float, 64><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 32:
        compute_hh_trafo_kernel_real<float, 32><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 16:
        compute_hh_trafo_kernel_real<float, 16><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 8:
        compute_hh_trafo_kernel_real<float, 8><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 4:
        compute_hh_trafo_kernel_real<float, 4><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 2:
        compute_hh_trafo_kernel_real<float, 2><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 1:
        compute_hh_trafo_kernel_real<float, 1><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    }

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("\n compute_hh_trafo CUDA kernel failed: %s \n",hipGetErrorString(err));
    }
}

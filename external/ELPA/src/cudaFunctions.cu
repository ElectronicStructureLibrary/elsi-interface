#include <stdio.h>
#include <math.h>
#include <stdio.h>
//    This file is part of ELPA.
//
//    The ELPA library was originally created by the ELPA consortium,
//    consisting of the following organizations:
//
//    - Max Planck Computing and Data Facility (MPCDF), formerly known as
//      Rechenzentrum Garching der Max-Planck-Gesellschaft (RZG),
//    - Bergische Universität Wuppertal, Lehrstuhl für angewandte
//      Informatik,
//    - Technische Universität München, Lehrstuhl für Informatik mit
//      Schwerpunkt Wissenschaftliches Rechnen ,
//    - Fritz-Haber-Institut, Berlin, Abt. Theorie,
//    - Max-Plack-Institut für Mathematik in den Naturwissenschaften,
//      Leipzig, Abt. Komplexe Strukutren in Biologie und Kognition,
//      and
//    - IBM Deutschland GmbH
//
//    This particular source code file contains additions, changes and
//    enhancements authored by Intel Corporation which is not part of
//    the ELPA consortium.
//
//    More information can be found here:
//    http://elpa.mpcdf.mpg.de/
//
//    ELPA is free software: you can redistribute it and/or modify
//    it under the terms of the version 3 of the license of the
//    GNU Lesser General Public License as published by the Free
//    Software Foundation.
//
//    ELPA is distributed in the hope that it will be useful,
//    but WITHOUT ANY WARRANTY; without even the implied warranty of
//    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//    GNU Lesser General Public License for more details.
//
//    You should have received a copy of the GNU Lesser General Public License
//    along with ELPA.  If not, see <http://www.gnu.org/licenses/>
//
//    ELPA reflects a substantial effort on the part of the original
//    ELPA consortium, and we ask you to respect the spirit of the
//    license that we chose: i.e., please contribute any changes you
//    may have back to the original ELPA library distribution, and keep
//    any derivatives of ELPA under the same license that we chose for
//    the original distribution, the GNU Lesser General Public License.
//
//
// --------------------------------------------------------------------------------------------------
//
// This file was written by A. Marek, MPCDF


#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <alloca.h>
#include <stdint.h>

#include "config-f90.h"

#define errormessage(x, ...) do { fprintf(stderr, "%s:%d " x, __FILE__, __LINE__, __VA_ARGS__ ); } while (0)

#ifdef DEBUG_CUDA
#define debugmessage(x, ...) do { fprintf(stderr, "%s:%d " x, __FILE__, __LINE__, __VA_ARGS__ ); } while (0)
#else
#define debugmessage(x, ...)
#endif

#ifdef WITH_GPU_VERSION
extern "C" {
  int cudaSetDeviceFromC(int n) {

    hipError_t cuerr = hipSetDevice(n);
    if (cuerr != hipSuccess) {
      errormessage("Error in hipSetDevice: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }

  int cudaGetDeviceCountFromC(int *count) {

    hipError_t cuerr = hipGetDeviceCount(count);
    if (cuerr != hipSuccess) {
      errormessage("Error in hipGetDeviceCount: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }

  int cudaDeviceSynchronizeFromC() {

    hipError_t cuerr = hipDeviceSynchronize();
    if (cuerr != hipSuccess) {
      errormessage("Error in hipGetDeviceCount: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }


  int cudaMallocFromC(intptr_t *a, size_t width_height) {

    hipError_t cuerr = hipMalloc((void **) a, width_height);
#ifdef DEBUG_CUDA
    printf("Malloc pointer address: %p \n", *a);
#endif
    if (cuerr != hipSuccess) {
      errormessage("Error in hipMalloc: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }
  int cudaFreeFromC(intptr_t *a) {
#ifdef DEBUG_CUDA
    printf("Free pointer address: %p \n", a);
#endif
    hipError_t cuerr = hipFree(a);

    if (cuerr != hipSuccess) {
      errormessage("Error in hipFree: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }

  int cudaMemsetFromC(intptr_t *a, int value, size_t count) {

    hipError_t cuerr = hipMemset( a, value, count);
    if (cuerr != hipSuccess) {
      errormessage("Error in hipMemset: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }

  int cudaMemcpyFromC(intptr_t *dest, intptr_t *src, size_t count, int dir) {

    hipError_t cuerr = hipMemcpy( dest, src, count, (hipMemcpyKind)dir);
    if (cuerr != hipSuccess) {
      errormessage("Error in hipMemcpy: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }

  int cudaMemcpy2dFromC(intptr_t *dest, size_t dpitch, intptr_t *src, size_t spitch, size_t width, size_t height, int dir) {

    hipError_t cuerr = hipMemcpy2D( dest, dpitch, src, spitch, width, height, (hipMemcpyKind)dir);
    if (cuerr != hipSuccess) {
      errormessage("Error in cudaMemcpy2d: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }
  int cudaMemcpyDeviceToDeviceFromC(void) {
      int val = hipMemcpyDeviceToDevice;
      return val;
  }
  int cudaMemcpyHostToDeviceFromC(void) {
      int val = hipMemcpyHostToDevice;
      return val;
  }
  int cudaMemcpyDeviceToHostFromC(void) {
      int val = hipMemcpyDeviceToHost;
      return val;
  }
  int cudaHostRegisterPortableFromC(void) {
      int val = hipHostRegisterPortable;
      return val;
  }
  int cudaHostRegisterMappedFromC(void) {
      int val = hipHostRegisterMapped;
      return val;
  }
}
#endif /* WITH_GPU_VERSION */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <alloca.h>
#include <stdint.h>
#include <complex.h>
#include <hipblas.h>

#define errormessage(x, ...) do { fprintf(stderr, "%s:%d " x, __FILE__, __LINE__, __VA_ARGS__ ); } while (0)

extern "C" {

    int cublasCreateFromC(intptr_t *cublas_handle) {
//     printf("in c: %p\n", *cublas_handle);
    *cublas_handle = (intptr_t) malloc(sizeof(hipblasHandle_t));
//     printf("in c: %p\n", *cublas_handle);
    hipblasStatus_t status = hipblasCreate((hipblasHandle_t*) *cublas_handle);
    if (status == HIPBLAS_STATUS_SUCCESS) {
//       printf("all OK\n");
      return 1;
    }
    else if (status == HIPBLAS_STATUS_NOT_INITIALIZED) {
      errormessage("Error in cublasCreate: %s\n", "the CUDA Runtime initialization failed");
      return 0;
    }
    else if (status == HIPBLAS_STATUS_ALLOC_FAILED) {
      errormessage("Error in cublasCreate: %s\n", "the resources could not be allocated");
      return 0;
    }
    else{
      errormessage("Error in cublasCreate: %s\n", "unknown error");
      return 0;
    }
  }

  int cublasDestroyFromC(intptr_t *cublas_handle) {
    hipblasStatus_t status = hipblasDestroy(*((hipblasHandle_t*) *cublas_handle));
    *cublas_handle = (intptr_t) NULL;
    if (status == HIPBLAS_STATUS_SUCCESS) {
//       printf("all OK\n");
      return 1;
    }
    else if (status == HIPBLAS_STATUS_NOT_INITIALIZED) {
      errormessage("Error in cublasDestroy: %s\n", "the library has not been initialized");
      return 0;
    }
    else{
      errormessage("Error in cublasCreate: %s\n", "unknown error");
      return 0;
    }
  }

  int cudaSetDeviceFromC(int n) {

    hipError_t cuerr = hipSetDevice(n);
    if (cuerr != hipSuccess) {
      errormessage("Error in cudaSetDevice: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }

  int cudaGetDeviceCountFromC(int *count) {

    hipError_t cuerr = hipGetDeviceCount(count);
    if (cuerr != hipSuccess) {
      errormessage("Error in cudaGetDeviceCount: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }

  int cudaDeviceSynchronizeFromC() {

    hipError_t cuerr = hipDeviceSynchronize();
    if (cuerr != hipSuccess) {
      errormessage("Error in cudaGetDeviceCount: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }


  int cudaMallocFromC(intptr_t *a, size_t width_height) {

    hipError_t cuerr = hipMalloc((void **) a, width_height);
    if (cuerr != hipSuccess) {
      errormessage("Error in cudaMalloc: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }
  int cudaFreeFromC(intptr_t *a) {
    hipError_t cuerr = hipFree(a);

    if (cuerr != hipSuccess) {
      errormessage("Error in cudaFree: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }

  int cudaMallocHostFromC(intptr_t *a, size_t width_height) {

    hipError_t cuerr = hipHostMalloc((void **) a, width_height, hipHostMallocDefault);
    if (cuerr != hipSuccess) {
      errormessage("Error in cudaMallocHost: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }
  int cudaFreeHostFromC(intptr_t *a) {
    hipError_t cuerr = hipHostFree(a);

    if (cuerr != hipSuccess) {
      errormessage("Error in cudaFreeHost: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }

  int cudaMemsetFromC(intptr_t *a, int value, size_t count) {

    hipError_t cuerr = hipMemset( a, value, count);
    if (cuerr != hipSuccess) {
      errormessage("Error in cudaMemset: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }

  int cudaMemcpyFromC(intptr_t *dest, intptr_t *src, size_t count, int dir) {

    hipError_t cuerr = hipMemcpy( dest, src, count, (hipMemcpyKind)dir);
    if (cuerr != hipSuccess) {
      errormessage("Error in cudaMemcpy: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }

  int cudaMemcpy2dFromC(intptr_t *dest, size_t dpitch, intptr_t *src, size_t spitch, size_t width, size_t height, int dir) {

    hipError_t cuerr = hipMemcpy2D( dest, dpitch, src, spitch, width, height, (hipMemcpyKind)dir);
    if (cuerr != hipSuccess) {
      errormessage("Error in cudaMemcpy2d: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }

  int cudaHostRegisterFromC(intptr_t *a, int value, int flag) {

    hipError_t cuerr = hipHostRegister( a, value, flag);
    if (cuerr != hipSuccess) {
      errormessage("Error in cudaHostRegister: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }

  int cudaHostUnregisterFromC(intptr_t *a) {

    hipError_t cuerr = hipHostUnregister( a);
    if (cuerr != hipSuccess) {
      errormessage("Error in cudaHostUnregister: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }

  int cudaMemcpyDeviceToDeviceFromC(void) {
      int val = hipMemcpyDeviceToDevice;
      return val;
  }
  int cudaMemcpyHostToDeviceFromC(void) {
      int val = hipMemcpyHostToDevice;
      return val;
  }
  int cudaMemcpyDeviceToHostFromC(void) {
      int val = hipMemcpyDeviceToHost;
      return val;
  }
  int cudaHostRegisterDefaultFromC(void) {
      int val = hipHostRegisterDefault;
      return val;
  }
  int cudaHostRegisterPortableFromC(void) {
      int val = hipHostRegisterPortable;
      return val;
  }
  int cudaHostRegisterMappedFromC(void) {
      int val = hipHostRegisterMapped;
      return val;
  }

  hipblasOperation_t operation_new_api(char trans) {
    if (trans == 'N' || trans == 'n') {
      return HIPBLAS_OP_N;
    }
    else if (trans == 'T' || trans == 't') {
      return HIPBLAS_OP_T;
    }
    else if (trans == 'C' || trans == 'c') {
      return HIPBLAS_OP_C;
    }
    else {
      errormessage("Error when transfering %c to cublasOperation_t\n",trans);
      // or abort?
      return HIPBLAS_OP_N;
    }
  }


  hipblasFillMode_t fill_mode_new_api(char uplo) {
    if (uplo == 'L' || uplo == 'l') {
      return HIPBLAS_FILL_MODE_LOWER;
    }
    else if(uplo == 'U' || uplo == 'u') {
      return HIPBLAS_FILL_MODE_UPPER;
    }
    else {
      errormessage("Error when transfering %c to cublasFillMode_t\n", uplo);
      // or abort?
      return HIPBLAS_FILL_MODE_LOWER;
    }
  }

  hipblasSideMode_t side_mode_new_api(char side) {
    if (side == 'L' || side == 'l') {
      return HIPBLAS_SIDE_LEFT;
    }
    else if (side == 'R' || side == 'r') {
      return HIPBLAS_SIDE_RIGHT;
    }
    else{
      errormessage("Error when transfering %c to cublasSideMode_t\n", side);
      // or abort?
      return HIPBLAS_SIDE_LEFT;
    }
  }

  hipblasDiagType_t diag_type_new_api(char diag) {
    if (diag == 'N' || diag == 'n') {
      return HIPBLAS_DIAG_NON_UNIT;
    }
    else if (diag == 'U' || diag == 'u') {
      return HIPBLAS_DIAG_UNIT;
    }
    else {
      errormessage("Error when transfering %c to cublasDiagMode_t\n", diag);
      // or abort?
      return HIPBLAS_DIAG_NON_UNIT;
    }
  }



  void cublasDgemv_elpa_wrapper (intptr_t handle, char trans, int m, int n, double alpha,
                               const double *A, int lda,  const double *x, int incx,
                               double beta, double *y, int incy) {

    hipblasDgemv(*((hipblasHandle_t*)handle), operation_new_api(trans),
                m, n, &alpha, A, lda, x, incx, &beta, y, incy);
  }

  void cublasSgemv_elpa_wrapper (intptr_t handle, char trans, int m, int n, float alpha,
                               const float *A, int lda,  const float *x, int incx,
                               float beta, float *y, int incy) {

    hipblasSgemv(*((hipblasHandle_t*)handle), operation_new_api(trans),
                m, n, &alpha, A, lda, x, incx, &beta, y, incy);
  }

  void cublasZgemv_elpa_wrapper (intptr_t handle, char trans, int m, int n, double _Complex alpha,
                               const double _Complex *A, int lda,  const double _Complex *x, int incx,
                               double _Complex beta, double _Complex *y, int incy) {

    hipDoubleComplex alpha_casted = *((hipDoubleComplex*)(&alpha));
    hipDoubleComplex beta_casted = *((hipDoubleComplex*)(&beta));

    const hipDoubleComplex* A_casted = (const hipDoubleComplex*) A;
    const hipDoubleComplex* x_casted = (const hipDoubleComplex*) x;
    hipDoubleComplex* y_casted = (hipDoubleComplex*) y;

    hipblasZgemv(*((hipblasHandle_t*)handle), operation_new_api(trans),
                m, n, &alpha_casted, A_casted, lda, x_casted, incx, &beta_casted, y_casted, incy);
  }

  void cublasCgemv_elpa_wrapper (intptr_t handle, char trans, int m, int n, float _Complex alpha,
                               const float _Complex *A, int lda,  const float _Complex *x, int incx,
                               float _Complex beta, float _Complex *y, int incy) {

    hipFloatComplex alpha_casted = *((hipFloatComplex*)(&alpha));
    hipFloatComplex beta_casted = *((hipFloatComplex*)(&beta));

    const hipFloatComplex* A_casted = (const hipFloatComplex*) A;
    const hipFloatComplex* x_casted = (const hipFloatComplex*) x;
    hipFloatComplex* y_casted = (hipFloatComplex*) y;

    hipblasCgemv(*((hipblasHandle_t*)handle), operation_new_api(trans),
                m, n, &alpha_casted, A_casted, lda, x_casted, incx, &beta_casted, y_casted, incy);
  }


  void cublasDgemm_elpa_wrapper (intptr_t handle, char transa, char transb, int m, int n, int k,
                               double alpha, const double *A, int lda,
                               const double *B, int ldb, double beta,
                               double *C, int ldc) {

    hipblasDgemm(*((hipblasHandle_t*)handle), operation_new_api(transa), operation_new_api(transb),
                m, n, k, &alpha, A, lda, B, ldb, &beta, C, ldc);
  }

  void cublasSgemm_elpa_wrapper (intptr_t handle, char transa, char transb, int m, int n, int k,
                               float alpha, const float *A, int lda,
                               const float *B, int ldb, float beta,
                               float *C, int ldc) {

    hipblasSgemm(*((hipblasHandle_t*)handle), operation_new_api(transa), operation_new_api(transb),
                m, n, k, &alpha, A, lda, B, ldb, &beta, C, ldc);
  }

  void cublasZgemm_elpa_wrapper (intptr_t handle, char transa, char transb, int m, int n, int k,
                               double _Complex alpha, const double _Complex *A, int lda,
                               const double _Complex *B, int ldb, double _Complex beta,
                               double _Complex *C, int ldc) {

    hipDoubleComplex alpha_casted = *((hipDoubleComplex*)(&alpha));
    hipDoubleComplex beta_casted = *((hipDoubleComplex*)(&beta));

    const hipDoubleComplex* A_casted = (const hipDoubleComplex*) A;
    const hipDoubleComplex* B_casted = (const hipDoubleComplex*) B;
    hipDoubleComplex* C_casted = (hipDoubleComplex*) C;

    hipblasZgemm(*((hipblasHandle_t*)handle), operation_new_api(transa), operation_new_api(transb),
                m, n, k, &alpha_casted, A_casted, lda, B_casted, ldb, &beta_casted, C_casted, ldc);
  }

  void cublasCgemm_elpa_wrapper (intptr_t handle, char transa, char transb, int m, int n, int k,
                               float _Complex alpha, const float _Complex *A, int lda,
                               const float _Complex *B, int ldb, float _Complex beta,
                               float _Complex *C, int ldc) {

    hipFloatComplex alpha_casted = *((hipFloatComplex*)(&alpha));
    hipFloatComplex beta_casted = *((hipFloatComplex*)(&beta));

    const hipFloatComplex* A_casted = (const hipFloatComplex*) A;
    const hipFloatComplex* B_casted = (const hipFloatComplex*) B;
    hipFloatComplex* C_casted = (hipFloatComplex*) C;

    hipblasCgemm(*((hipblasHandle_t*)handle), operation_new_api(transa), operation_new_api(transb),
                m, n, k, &alpha_casted, A_casted, lda, B_casted, ldb, &beta_casted, C_casted, ldc);
  }


  // todo: new CUBLAS API diverged from standard BLAS api for these functions
  // todo: it provides out-of-place (and apparently more efficient) implementation
  // todo: by passing B twice (in place of C as well), we should fall back to in-place algorithm

  void cublasDtrmm_elpa_wrapper (intptr_t handle, char side, char uplo, char transa, char diag,
                               int m, int n, double alpha, const double *A,
                               int lda, double *B, int ldb){

    hipblasDtrmm(*((hipblasHandle_t*)handle), side_mode_new_api(side), fill_mode_new_api(uplo), operation_new_api(transa),
                diag_type_new_api(diag), m, n, &alpha, A, lda, B, ldb, B, ldb);
  }

  void cublasStrmm_elpa_wrapper (intptr_t handle, char side, char uplo, char transa, char diag,
                               int m, int n, float alpha, const float *A,
                               int lda, float *B, int ldb){

    hipblasStrmm(*((hipblasHandle_t*)handle), side_mode_new_api(side), fill_mode_new_api(uplo), operation_new_api(transa),
                diag_type_new_api(diag), m, n, &alpha, A, lda, B, ldb, B, ldb);
  }

  void cublasZtrmm_elpa_wrapper (intptr_t handle, char side, char uplo, char transa, char diag,
                               int m, int n, double _Complex alpha, const double _Complex *A,
                               int lda, double _Complex *B, int ldb){

    hipDoubleComplex alpha_casted = *((hipDoubleComplex*)(&alpha));

    const hipDoubleComplex* A_casted = (const hipDoubleComplex*) A;
    hipDoubleComplex* B_casted = (hipDoubleComplex*) B;

    hipblasZtrmm(*((hipblasHandle_t*)handle), side_mode_new_api(side), fill_mode_new_api(uplo), operation_new_api(transa),
                diag_type_new_api(diag), m, n, &alpha_casted, A_casted, lda, B_casted, ldb, B_casted, ldb);
  }

  void cublasCtrmm_elpa_wrapper (intptr_t handle, char side, char uplo, char transa, char diag,
                               int m, int n, float _Complex alpha, const float _Complex *A,
                               int lda, float _Complex *B, int ldb){

    hipFloatComplex alpha_casted = *((hipFloatComplex*)(&alpha));

    const hipFloatComplex* A_casted = (const hipFloatComplex*) A;
    hipFloatComplex* B_casted = (hipFloatComplex*) B;

    hipblasCtrmm(*((hipblasHandle_t*)handle), side_mode_new_api(side), fill_mode_new_api(uplo), operation_new_api(transa),
                diag_type_new_api(diag), m, n, &alpha_casted, A_casted, lda, B_casted, ldb, B_casted, ldb);
  }


}

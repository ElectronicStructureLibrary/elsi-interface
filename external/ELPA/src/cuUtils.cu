#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_complex.h>

// Reset a reduction block
// Limitation: the thread-block size must be a divider of the reduction block's size
__device__ void reset_shared_block_c ( double * s_block, int b_size)
{
    int i, t_idx, s_chunk ;
    t_idx = threadIdx.x;
    s_chunk = b_size / blockDim.x;
    for(i = ((t_idx - 1) * s_chunk + 1) ; i < (t_idx * s_chunk); i++)
        s_block[i] = 0.0 ;
    __syncthreads();
}

// Reset 2 reduction blocks without an explicit synchronization at the end
// Limitation: : the thread-block size must be a divider of the reduction block's size
__device__ void reset_shared_block_pair_c( double *s_block_1, double *s_block_2, int b_size)
{
    int i, t_idx, s_chunk;

    t_idx = threadIdx.x;
    s_chunk = b_size / blockDim.x;
    for(i = ((t_idx - 1) * s_chunk + 1); i < (t_idx * s_chunk); i++)
    {    s_block_1[i] = 0.0 ;
        s_block_2[i] = 0.0 ;
    }
}
// Reset a reduction block
// Limitation: the thread-block size must be a divider of the reduction block's size
__device__ void reset_shared_block_c_complex ( hipDoubleComplex * s_block, int b_size)
{
    int i, t_idx, s_chunk ;
    t_idx = threadIdx.x;
    s_chunk = b_size / blockDim.x;
    for(i = ((t_idx - 1) * s_chunk + 1) ; i < (t_idx * s_chunk); i++)
       { s_block[i].x = 0.0 ;
        s_block[i].y = 0.0 ;}
    __syncthreads();
}

// Reset 2 reduction blocks without an explicit synchronization at the end
// Limitation: : the thread-block size must be a divider of the reduction block's size
__device__ void reset_shared_block_pair_c_complex( hipDoubleComplex *s_block_1, hipDoubleComplex *s_block_2, int b_size)
{
    int i, t_idx, s_chunk;

    t_idx = threadIdx.x;
    s_chunk = b_size / blockDim.x;
    for(i = ((t_idx - 1) * s_chunk + 1); i < (t_idx * s_chunk); i++)
    {    s_block_1[i].x = 0.0 ;
        s_block_2[i].x= 0.0 ;
        s_block_1[i].y = 0.0 ;
        s_block_2[i].y= 0.0 ;
    }
}

__device__ void warp_reduce_complex( hipDoubleComplex *s_block)
{
    int t_idx ;
    t_idx = threadIdx.x;
    __syncthreads();

	if (t_idx < 32)
        {

        s_block[t_idx] = hipCadd(hipCadd(s_block[t_idx],s_block[t_idx + 32]) , hipCadd( s_block[t_idx + 64], s_block[t_idx + 96]) );
        if (t_idx < 8)
        {
        s_block[t_idx] = hipCadd(hipCadd(s_block[t_idx],s_block[t_idx + 8] ) , hipCadd( s_block[t_idx + 16] , s_block[t_idx + 24] ) );

        }
        if (t_idx < 4)
        {
        s_block[t_idx] = hipCadd(s_block[t_idx] , s_block[t_idx + 4]) ;
        }
        if (t_idx < 1)
        {
        s_block[t_idx] = hipCadd(hipCadd(s_block[t_idx],s_block[t_idx + 1] ) , hipCadd( s_block[t_idx +2] , s_block[t_idx + 3] ) );
        }
        }

}

__global__ void my_pack_c_kernel_complex(const int n_offset, const int max_idx, const int stripe_width, const int a_dim2, const int stripe_count, const int l_nev, hipDoubleComplex* src, hipDoubleComplex* dst)
{
    int b_id, t_id ;
    int dst_ind ;
    b_id = blockIdx.y;
    t_id = threadIdx.x;

    dst_ind = b_id * stripe_width + t_id;
    if (dst_ind < max_idx)
    {
        // dimension of dst - lnev, nblk
        // dimension of src - stripe_width,a_dim2,stripe_count
	dst[dst_ind + (l_nev*blockIdx.x)].x = src[t_id + (stripe_width*(n_offset + blockIdx.x)) + ( b_id *stripe_width*a_dim2)].x;
        dst[dst_ind + (l_nev*blockIdx.x)].y = src[t_id + (stripe_width*(n_offset + blockIdx.x)) + ( b_id *stripe_width*a_dim2)].y;
     }

}
__global__ void  my_unpack_c_kernel_complex( const int n_offset, const int max_idx, const int stripe_width, const int a_dim2, const int stripe_count, const int l_nev, hipDoubleComplex* src, hipDoubleComplex* dst)
{
    int b_id, t_id ;
    int src_ind;

    b_id = blockIdx.y;
    t_id = threadIdx.x;

    src_ind = b_id * stripe_width + t_id;
    if (src_ind < max_idx)
{	dst[ t_id + ((n_offset + blockIdx.x) * stripe_width) + (b_id * stripe_width * a_dim2 )].x = src[ src_ind  + (blockIdx.x) *l_nev ].x;
	dst[ t_id + ((n_offset + blockIdx.x) * stripe_width) + (b_id * stripe_width * a_dim2 )].y = src[ src_ind  + (blockIdx.x) *l_nev ].y;
}
}


__global__ void extract_hh_tau_c_kernel_complex(hipDoubleComplex* hh, hipDoubleComplex* hh_tau, const int nbw, const int n, int val)
{
    int h_idx ;

    h_idx = (blockIdx.x) * blockDim.x + threadIdx.x;

    if (h_idx < n)
    {
        //dimension of hh - (nbw, max_blk_size)
        //dimension of hh_tau - max_blk_size
        hh_tau[h_idx] = hh[h_idx * nbw] ;
        //  Replace the first element in the HH reflector with 1.0 or 0.0
        if( val == 0)
        {
         hh[(h_idx * nbw)].x = 1.0;
	 hh[h_idx *nbw].y= 0.0;
        }
        else
        {
        hh[(h_idx * nbw)].x = 0.0;
	hh[h_idx*nbw].y =0.0;
        }
     }
}

__global__ void  compute_hh_dotp_c_kernel_complex(hipDoubleComplex* hh, hipDoubleComplex* v_dot, const int nbw, const int n)
{
   __shared__ hipDoubleComplex hh_s[128] ;

    int t_idx, v_idx;

    //  The vector index (v_idx) identifies the pair of HH reflectors from which the dot product is computed
    v_idx = blockIdx.x  ;

    //  The thread index indicates the position within the two HH reflectors
    t_idx = threadIdx.x ;

    if (t_idx  > 0)
    { 

       hh_s[t_idx] = hipCmul(hipConj(hh[t_idx + v_idx * nbw]),   hh[ (t_idx - 1) +  (v_idx +1)* nbw]) ;
    }
    else
    {   
        hh_s[t_idx].x = 0.0 ;
        hh_s[t_idx].y = 0.0;
    } 

  //  Compute the dot product using a fast reduction
     warp_reduce_complex(hh_s);
     __syncthreads();

      if(t_idx == 0)
       {
	v_dot[v_idx] = hh_s[0] ;
	}

}

extern "C" void launch_my_pack_c_kernel_complex(const int row_count, const int n_offset, const int max_idx, const int stripe_width, const int a_dim2, const int stripe_count, const int l_nev, hipDoubleComplex* a_dev, hipDoubleComplex* row_group_dev)
{
	if(stripe_width == 0) return;
        dim3  grid_size;
        grid_size = dim3(row_count, stripe_count, 1);
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) printf("error prior to mypack kernel: %s, %d\n",hipGetErrorString(err), err);
        my_pack_c_kernel_complex<<<grid_size, stripe_width>>>(n_offset, max_idx, stripe_width, a_dim2, stripe_count, l_nev, a_dev, row_group_dev);
        err = hipGetLastError();
        if ( err!= hipSuccess)
        {
                printf("\n my pack_kernel failed  %s \n",hipGetErrorString(err) );
        }
}

extern "C" void launch_compute_hh_dotp_c_kernel_complex(hipDoubleComplex* bcast_buffer_dev, hipDoubleComplex* hh_dot_dev,const int nbw,const int n)
{
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) printf("error prior to compute_hh kernel: %s, %d\n",hipGetErrorString(err), err);
        compute_hh_dotp_c_kernel_complex<<< n-1, nbw >>>(bcast_buffer_dev, hh_dot_dev, nbw, n);

        err = hipGetLastError();
        if ( err!= hipSuccess)
        {
                printf("\n compute _kernel failed  %s \n",hipGetErrorString(err) );
        }
}

extern "C" void launch_extract_hh_tau_c_kernel_complex(hipDoubleComplex* bcast_buffer_dev, hipDoubleComplex* hh_tau_dev, const int nbw, const int n , const int is_zero)
{
        int grid_size;
        grid_size = 1 + (n - 1) / 256;
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) printf("error prior to extract kernel: %s, %d\n",hipGetErrorString(err), err);
        extract_hh_tau_c_kernel_complex<<<grid_size,256>>>(bcast_buffer_dev,hh_tau_dev, nbw, n, is_zero);
        err = hipGetLastError();
        if ( err!= hipSuccess)
        {
                printf("\n  extract _kernel failed  %s \n",hipGetErrorString(err) );
        }

}

extern "C" void launch_my_unpack_c_kernel_complex( const int row_count, const int n_offset, const int max_idx, const int stripe_width,const int a_dim2, const int stripe_count, const int l_nev, hipDoubleComplex* row_group_dev, hipDoubleComplex* a_dev)
{
	if(stripe_width == 0) return;
        dim3  grid_size;
        grid_size = dim3(row_count, stripe_count, 1);
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) printf("error prior to unpack kernel: %s, %d\n",hipGetErrorString(err), err);
        my_unpack_c_kernel_complex<<<grid_size, stripe_width>>>(n_offset, max_idx, stripe_width, a_dim2, stripe_count, l_nev, row_group_dev , a_dev);
        err = hipGetLastError();
        if ( err != hipSuccess)
        {
            printf("\n  my_unpack_c_kernel failed  %s \n",hipGetErrorString(err) );
        }
}

__device__ void warp_reduce_c( double *s_block)
{
    int t_idx ;
    t_idx = threadIdx.x;
    __syncthreads();

        if (t_idx < 32)
	{
                s_block[t_idx] = s_block[t_idx] + s_block[t_idx + 32] + s_block[t_idx + 64] + s_block[t_idx + 96] ;
        if (t_idx < 8)
                s_block[t_idx] = s_block[t_idx] + s_block[t_idx + 8] + s_block[t_idx + 16] + s_block[t_idx + 24];
        if (t_idx < 4)
                s_block[t_idx] = s_block[t_idx] + s_block[t_idx + 4];
        if (t_idx < 1)
                s_block[t_idx] = s_block[t_idx] + s_block[t_idx + 1] + s_block[t_idx + 2] + s_block[t_idx + 3];
	}
}

__global__ void my_pack_c_kernel(const int n_offset, const int max_idx, const int stripe_width, const int a_dim2, const int stripe_count, const int l_nev, double* src, double* dst)
{    
    int b_id, t_id ;
    int dst_ind ;
    b_id = blockIdx.y;
    t_id = threadIdx.x;

    dst_ind = b_id * stripe_width + t_id;
    if (dst_ind < max_idx)
    {
	// dimension of dst - lnev, nblk
	// dimension of src - stripe_width,a_dim2,stripe_count
    	*(dst + dst_ind + (l_nev*blockIdx.x) ) = *(src + t_id + (stripe_width*(n_offset + blockIdx.x)) + ( b_id *stripe_width*a_dim2 ));
     }

}

__global__ void  my_unpack_c_kernel( const int n_offset, const int max_idx, const int stripe_width, const int a_dim2, const int stripe_count, const int l_nev, double* src, double* dst)
{
    int b_id, t_id ;
    int src_ind;
   
    b_id = blockIdx.y;
    t_id = threadIdx.x;

    src_ind = b_id * stripe_width + t_id;
    if (src_ind < max_idx)
	*(dst + (t_id + ((n_offset + blockIdx.x) * stripe_width) + (b_id * stripe_width * a_dim2 ))) = *(src + src_ind  + (blockIdx.x) *l_nev );

}
__global__ void compute_kernel_reduce( hipDoubleComplex* a_dev, int lda , int n ,int nbw ,  hipDoubleComplex *h1_dev )
{
    int  t_id ;
    int st_ind;

    t_id = threadIdx.x;
    
    st_ind = (t_id*(t_id+1))/2;
    if(t_id< n)
    {     
	for(int i =0;i<=t_id;i++)
        { 
         h1_dev[st_ind + i] = a_dev[t_id *lda + i ] ;
	}
    }
    __syncthreads();


}
__global__ void compute_kernel_reduce_1( hipDoubleComplex* a_dev, int lda , int n, hipDoubleComplex *h1_dev )
{
    int  t_id ;
    int st_ind;

    t_id = threadIdx.x;

    st_ind = (t_id*(t_id+1))/2;
    if(t_id< n)
    {
        for(int i =0;i<=t_id;i++)
         {
	  a_dev[t_id *lda + i ] = h1_dev[st_ind + i];
	  a_dev[ (i-1)*lda + t_id ] = hipConj(a_dev[ t_id *lda + i-1]) ;
	} 
    }
    __syncthreads();


}

__global__ void  dot_product_c_kernel( hipDoubleComplex* hs_dev, hipDoubleComplex* hv_new_dev, hipDoubleComplex tau_new_dev, hipDoubleComplex*  x_dev, hipDoubleComplex *h_dev, hipDoubleComplex *hv_dev, int nr)
{
    int t_id ;
  
    __shared__ hipDoubleComplex x_dev_temp[128];
    __shared__ hipDoubleComplex x_val;

    //b_id = blockIdx.y;
    t_id = threadIdx.x;
 
    if(t_id<nr)
	 x_dev_temp[t_id] = hipCmul( hipConj(hs_dev[t_id]), hv_new_dev[t_id]) ; 
    __syncthreads();

    if(t_id==0)
    {
        for(int i=1;i<nr;i++)
	x_dev_temp[t_id] = hipCadd(x_dev_temp[t_id],x_dev_temp[t_id +i]);
    }
    __syncthreads();
     if(t_id ==0)
    {
      x_val =  hipCmul(x_dev_temp[t_id], tau_new_dev);
      x_dev[0] = x_val;
    }
	__syncthreads();
}


__global__ void  dot_product_c_kernel_1(   hipDoubleComplex*  ab_dev, hipDoubleComplex *hs_dev,  hipDoubleComplex*  hv_new_dev, hipDoubleComplex*  x_dev, hipDoubleComplex *h_dev, hipDoubleComplex *hv_dev,  int nb, int nr , int ns )
{
    int t_id = threadIdx.x;
        int i; 
 
    if((t_id>0 )&& (t_id < nb))
    {
	h_dev[t_id] = hipCsub(h_dev[t_id], hipCmul(x_dev[0],hv_dev[t_id]));
        for(i=0;i<nr;i++)
	{
	 ab_dev[ i+nb-t_id + (t_id+ns-1)*2*nb ] = hipCsub(hipCsub(ab_dev[ i+nb-t_id + (t_id+ns-1)*2*nb],hipCmul(hv_new_dev[i],hipConj(h_dev[t_id])) ),hipCmul(hs_dev[i], hipConj(hv_dev[t_id])));
 	}
    }
 
   __syncthreads();

}
__global__ void  double_hh_transform_kernel( hipDoubleComplex*  ab_dev, hipDoubleComplex *hs_dev, hipDoubleComplex *hv_dev,  int nb,  int ns )
{
    int t_id = threadIdx.x;
    if((t_id>0 )&& (t_id < nb))
    {
         ab_dev[ nb-t_id + (t_id+ns-1)*2*nb ] = hipCsub(ab_dev[ nb-t_id + (t_id+ns-1)*2*nb],hipCmul(hs_dev[0], hipConj(hv_dev[t_id])));
        
    }

   __syncthreads();

}
__global__ void  double_hh_transform_kernel_2( hipDoubleComplex*  ab_dev, hipDoubleComplex *hd_dev, hipDoubleComplex *hv_dev,  int nc,  int ns , int nb )
{
    int t_id = threadIdx.x;
    if(t_id < nc) 
    {

         ab_dev[ t_id + (ns-1)*2*nb ] = hipCsub(hipCsub(ab_dev[ t_id + (ns-1)*2*nb],hipCmul(hd_dev[ t_id], hipConj(hv_dev[0]))) , hipCmul(hv_dev[ t_id], hipConj(hd_dev[0])));

    }

   __syncthreads();

}






__global__ void extract_hh_tau_c_kernel(double* hh, double* hh_tau, const int nbw, const int n, int val)
{
    int h_idx ;
    h_idx = (blockIdx.x) * blockDim.x + threadIdx.x;

    if (h_idx < n)
    {
	//dimension of hh - (nbw, max_blk_size)
	//dimension of hh_tau - max_blk_size 
        *(hh_tau + h_idx ) = *(hh +  (h_idx * nbw)) ;
        //  Replace the first element in the HH reflector with 1.0 or 0.0
	if( val == 0)
        *(hh + (h_idx * nbw)) = 1.0;
	else
	*(hh + (h_idx * nbw)) = 0.0;
     }
}

__global__ void  compute_hh_dotp_c_kernel(double* hh, double* v_dot, const int nbw, const int n)
{

   __shared__ double hh_s[128] ;

    int t_idx, v_idx;

    //  The vector index (v_idx) identifies the pair of HH reflectors from which the dot product is computed
    v_idx = blockIdx.x  ;

    //  The thread index indicates the position within the two HH reflectors
    t_idx = threadIdx.x ;

    //  The contents of the shared memory must be fully reset
    // reset_shared_block_c(hh_s, 128);

    //  Initialize the contents of the shared buffer (preparing for reduction)
    if (t_idx  > 0)
        *(hh_s + t_idx) = *(hh + t_idx + v_idx * nbw ) *  (*(hh + (t_idx - 1) +  (v_idx +1)* nbw)) ;
    else
        *(hh_s + t_idx) = 0.0 ;
    
     //  Compute the dot product using a fast reduction
     warp_reduce_c(hh_s); 

      if(t_idx == 0) 
      *(v_dot + v_idx) = *(hh_s) ;

}

extern "C" void launch_my_pack_c_kernel(const int row_count, const int n_offset, const int max_idx, const int stripe_width, const int a_dim2, const int stripe_count, const int l_nev, double* a_dev, double* row_group_dev)
{

	if(stripe_width == 0) return;
	dim3  grid_size;
        grid_size = dim3(row_count, stripe_count, 1);
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) printf("error prior to mypack kernel: %s, %d\n",hipGetErrorString(err), err);

	my_pack_c_kernel<<<grid_size, stripe_width>>>(n_offset, max_idx, stripe_width, a_dim2, stripe_count, l_nev, a_dev, row_group_dev);
	 err = hipGetLastError();
        if ( err!= hipSuccess)
        {
                printf("\n my pack_kernel failed  %s \n",hipGetErrorString(err) );
        }

}

extern "C" void launch_compute_hh_dotp_c_kernel(double* bcast_buffer_dev, double* hh_dot_dev,const int nbw,const int n)
{
	hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) printf("error prior to compute_hh kernel: %s, %d\n",hipGetErrorString(err), err);
        compute_hh_dotp_c_kernel<<< n-1, nbw >>>(bcast_buffer_dev, hh_dot_dev, nbw, n); 
	err = hipGetLastError();
        if ( err!= hipSuccess)
        {
                printf("\n compute _kernel failed  %s \n",hipGetErrorString(err) );
        }

	

}

extern "C" void launch_extract_hh_tau_c_kernel(double* bcast_buffer_dev, double* hh_tau_dev, const int nbw, const int n , const int is_zero)
{
	int grid_size;
	grid_size = 1 + (n - 1) / 256;
	hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) printf("error prior to extract kernel: %s, %d\n",hipGetErrorString(err), err);
	extract_hh_tau_c_kernel<<<grid_size,256>>>(bcast_buffer_dev,hh_tau_dev, nbw, n, is_zero);
	err = hipGetLastError();
	if ( err!= hipSuccess)
       	{
		printf("\n  extract _kernel failed  %s \n",hipGetErrorString(err) ); 
        }

}

extern "C" void launch_my_unpack_c_kernel( const int row_count, const int n_offset, const int max_idx, const int stripe_width,const int a_dim2, const int stripe_count, const int l_nev, double* row_group_dev, double* a_dev)
{

	if(stripe_width == 0) return;
        dim3  grid_size;
	grid_size = dim3(row_count, stripe_count, 1);
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) printf("error prior to unpack kernel: %s, %d\n",hipGetErrorString(err), err); 
        my_unpack_c_kernel<<<grid_size, stripe_width>>>(n_offset, max_idx, stripe_width, a_dim2, stripe_count, l_nev, row_group_dev , a_dev);
        err = hipGetLastError();
        if ( err != hipSuccess)
        {
	    printf("\n  my_unpack_c_kernel failed  %s \n",hipGetErrorString(err) );
        }
}
extern "C" void launch_dot_product_kernel( hipDoubleComplex* hs_dev, hipDoubleComplex* hv_new_dev, hipDoubleComplex tau_new_dev, hipDoubleComplex*  x_dev, hipDoubleComplex*  h_dev ,hipDoubleComplex*  hv_dev,int nr )
{

        dim3  grid_size;
        grid_size = dim3(1,1, 1);
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) printf("error prior to launch_dot_product kernel: %s, %d\n",hipGetErrorString(err), err);
        dot_product_c_kernel<<<grid_size, nr>>>(hs_dev, hv_new_dev, tau_new_dev, x_dev, h_dev, hv_dev, nr );
        err = hipGetLastError();
        if ( err != hipSuccess)
        {
            printf("\n dot product kernel failed  %s \n",hipGetErrorString(err) );

        }

}

extern "C" void launch_dot_product_kernel_1(  hipDoubleComplex*  ab_dev, hipDoubleComplex *hs_dev,  hipDoubleComplex*  hv_new_dev,hipDoubleComplex*  x_dev, hipDoubleComplex*  h_dev ,hipDoubleComplex*  hv_dev, int nb ,int nr , int ns)
{
        dim3  grid_size;
        grid_size = dim3(1,1, 1);
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) printf("error prior to launch_dot_product kernel: %s, %d\n",hipGetErrorString(err), err);
        dot_product_c_kernel_1<<<grid_size, nb>>>( ab_dev, hs_dev, hv_new_dev, x_dev, h_dev, hv_dev, nb, nr, ns );
        err = hipGetLastError();
        if ( err != hipSuccess)
        {
            printf("\n dot product kernel failed  %s \n",hipGetErrorString(err) );

        }

}


extern "C" void launch_dot_product_kernel_2(  hipDoubleComplex*  ab_dev, hipDoubleComplex *hs_dev,  hipDoubleComplex*  hv_dev,hipDoubleComplex*  hd_dev, int nb ,int nr , int ne)
{
        dim3  grid_size;
        grid_size = dim3(1,1, 1);
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) printf("error prior to launch_dot_product kernel: %s, %d\n",hipGetErrorString(err), err);
        err = hipGetLastError();
        if ( err != hipSuccess)
        {
            printf("\n dot product kernel failed  %s \n",hipGetErrorString(err) );

        }

}

extern "C" void launch_double_hh_transform_1( hipDoubleComplex*  ab_dev, hipDoubleComplex *hs_dev,hipDoubleComplex*  hv_dev, int nb , int ns)
{
        dim3  grid_size;
        grid_size = dim3(1,1, 1);
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) printf("error prior to launch_double_hh_transform kernel: %s, %d\n",hipGetErrorString(err), err);
        double_hh_transform_kernel<<<grid_size, nb>>>( ab_dev, hs_dev, hv_dev, nb,  ns );
        err = hipGetLastError();
        if ( err != hipSuccess)
        {
            printf("\n dot product kernel failed  %s \n",hipGetErrorString(err) );

        }

}

extern "C" void launch_double_hh_transform_2( hipDoubleComplex*  ab_dev, hipDoubleComplex *hd_dev,hipDoubleComplex*  hv_dev, int nc , int ns , int nb )
{
        dim3  grid_size;
        grid_size = dim3(1,1, 1);
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) printf("error prior to launch_double_hh_transform kernel: %s, %d\n",hipGetErrorString(err), err);
        double_hh_transform_kernel_2<<<grid_size, nc>>>( ab_dev, hd_dev, hv_dev, nc,  ns, nb );
        err = hipGetLastError();
        if ( err != hipSuccess)
        {
            printf("\n dot product kernel failed  %s \n",hipGetErrorString(err) );

        }

}



extern "C" void launch_compute_kernel_reduce( hipDoubleComplex* a_dev, int lda, int n,int nbw, hipDoubleComplex* h_dev)
{

        dim3  grid_size;
        grid_size = dim3(1,1, 1);
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) printf("error prior to launch_dot_product kernel: %s, %d\n",hipGetErrorString(err), err);
        compute_kernel_reduce<<<grid_size,n>>>(a_dev, lda, n, nbw,h_dev);
	hipDeviceSynchronize();
        err = hipGetLastError();
        if ( err != hipSuccess)
        {
            printf("\n dot product kernel failed  %s \n",hipGetErrorString(err) );

        }

}

extern "C" void launch_compute_kernel_reduce_1( hipDoubleComplex* a_dev, int lda, int n , hipDoubleComplex* h_dev)
{

        dim3  grid_size;
        grid_size = dim3(1,1, 1);
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) printf("error prior to launch_dot_product kernel: %s, %d\n",hipGetErrorString(err), err);
        compute_kernel_reduce_1<<<grid_size,n>>>(a_dev, lda, n, h_dev);
	hipDeviceSynchronize();
        err = hipGetLastError();
        if ( err != hipSuccess)
        {
            printf("\n dot product kernel failed  %s \n",hipGetErrorString(err) );

        }

}


extern "C" int cuda_MemcpyDeviceToDevice(int val)
{
      val = hipMemcpyDeviceToDevice;
      return val;
}

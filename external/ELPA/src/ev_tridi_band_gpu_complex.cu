//    This file is part of ELPA.
//
//    The ELPA library was originally created by the ELPA consortium,
//    consisting of the following organizations:
//
//    - Max Planck Computing and Data Facility (MPCDF), formerly known as
//      Rechenzentrum Garching der Max-Planck-Gesellschaft (RZG),
//    - Bergische Universität Wuppertal, Lehrstuhl für angewandte
//      Informatik,
//    - Technische Universität München, Lehrstuhl für Informatik mit
//      Schwerpunkt Wissenschaftliches Rechnen ,
//    - Fritz-Haber-Institut, Berlin, Abt. Theorie,
//    - Max-Plack-Institut für Mathematik in den Naturwissenschaften,
//      Leipzig, Abt. Komplexe Strukutren in Biologie und Kognition,
//      and
//    - IBM Deutschland GmbH
//
//    This particular source code file contains additions, changes and
//    enhancements authored by Intel Corporation which is not part of
//    the ELPA consortium.
//
//    More information can be found here:
//    http://elpa.mpcdf.mpg.de/
//
//    ELPA is free software: you can redistribute it and/or modify
//    it under the terms of the version 3 of the license of the
//    GNU Lesser General Public License as published by the Free
//    Software Foundation.
//
//    ELPA is distributed in the hope that it will be useful,
//    but WITHOUT ANY WARRANTY; without even the implied warranty of
//    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//    GNU Lesser General Public License for more details.
//
//    You should have received a copy of the GNU Lesser General Public License
//    along with ELPA.  If not, see <http://www.gnu.org/licenses/>
//
//    ELPA reflects a substantial effort on the part of the original
//    ELPA consortium, and we ask you to respect the spirit of the
//    license that we chose: i.e., please contribute any changes you
//    may have back to the original ELPA library distribution, and keep
//    any derivatives of ELPA under the same license that we chose for
//    the original distribution, the GNU Lesser General Public License.
//
//

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

template <typename T, unsigned int blk> __device__ void warp_reduce_complex(volatile T *s_block)
{
    unsigned int tid = threadIdx.x;

    if (blk >= 64)
    {
        if (tid < 32)
        {
            s_block[tid].x += s_block[tid + 32].x;
            s_block[tid].y += s_block[tid + 32].y;
        }
    }

    if (blk >= 32)
    {
        if (tid < 16)
        {
            s_block[tid].x += s_block[tid + 16].x;
            s_block[tid].y += s_block[tid + 16].y;
        }
    }

    if (blk >= 16)
    {
        if (tid < 8)
        {
            s_block[tid].x += s_block[tid + 8].x;
            s_block[tid].y += s_block[tid + 8].y;
        }
    }

    if (blk >= 8)
    {
        if (tid < 4)
        {
            s_block[tid].x += s_block[tid + 4].x;
            s_block[tid].y += s_block[tid + 4].y;
        }
    }

    if (blk >= 4)
    {
        if (tid < 2)
        {
            s_block[tid].x += s_block[tid + 2].x;
            s_block[tid].y += s_block[tid + 2].y;
        }
    }

    if (blk >= 2)
    {
        if (tid < 1)
        {
            s_block[tid].x += s_block[tid + 1].x;
            s_block[tid].y += s_block[tid + 1].y;
        }
    }
}

template <typename T, unsigned int blk> __device__ void reduce_complex(T *s_block)
{
    unsigned int tid = threadIdx.x;

    if (blk >= 1024)
    {
        if (tid < 512)
        {
            s_block[tid].x += s_block[tid + 512].x;
            s_block[tid].y += s_block[tid + 512].y;
        }

        __syncthreads();
    }

    if (blk >= 512)
    {
        if (tid < 256)
        {
            s_block[tid].x += s_block[tid + 256].x;
            s_block[tid].y += s_block[tid + 256].y;
        }

        __syncthreads();
    }

    if (blk >= 256)
    {
        if (tid < 128)
        {
            s_block[tid].x += s_block[tid + 128].x;
            s_block[tid].y += s_block[tid + 128].y;
        }

        __syncthreads();
    }

    if (blk >= 128)
    {
        if (tid < 64)
        {
            s_block[tid].x += s_block[tid + 64].x;
            s_block[tid].y += s_block[tid + 64].y;
        }

        __syncthreads();
    }

    if (tid < 32)
    {
        warp_reduce_complex<T, blk>(s_block);
    }

}

template <unsigned int blk>
__global__ void compute_hh_trafo_kernel_complex_double(hipDoubleComplex * __restrict__ q, const hipDoubleComplex * __restrict__ hh, const hipDoubleComplex * __restrict__ hh_tau, const int nb, const int ldq, const int ncols)
{
    __shared__ hipDoubleComplex q_s[blk + 1];
    __shared__ hipDoubleComplex dotp_s[blk];

    hipDoubleComplex q_v2;

    int q_off, h_off, j;

    unsigned int tid = threadIdx.x;
    unsigned int bid = blockIdx.x;

    j = ncols;
    q_off = bid + (j + tid) * ldq;
    h_off = tid + j * nb;
    q_s[tid] = q[q_off];

    while (j >= 1)
    {
        if (tid == 0)
        {
            q_s[tid] = q[q_off];
        }

        q_v2 = q_s[tid];
        dotp_s[tid] = hipCmul(q_v2, hipConj(hh[h_off]));

        __syncthreads();

        reduce_complex<hipDoubleComplex, blk>(dotp_s);

        __syncthreads();

        q_v2 = hipCsub(q_v2, hipCmul(hipCmul(dotp_s[0], hh_tau[j]), hh[h_off]));
        q_s[tid + 1] = q_v2;

        if ((j == 1) || (tid == blockDim.x - 1))
        {
            q[q_off] = q_v2;
        }

        __syncthreads();

        q_off -= ldq;
        h_off -= nb;
        j -= 1;
    }
}

extern "C" void launch_compute_hh_trafo_c_kernel_complex_double(hipDoubleComplex *q, const hipDoubleComplex *hh, const hipDoubleComplex *hh_tau, const int nev, const int nb, const int ldq, const int ncols)
{
    hipError_t err;

    switch (nb)
    {
    case 1024:
        compute_hh_trafo_kernel_complex_double<1024><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 512:
        compute_hh_trafo_kernel_complex_double<512><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 256:
        compute_hh_trafo_kernel_complex_double<256><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 128:
        compute_hh_trafo_kernel_complex_double<128><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 64:
        compute_hh_trafo_kernel_complex_double<64><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 32:
        compute_hh_trafo_kernel_complex_double<32><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 16:
        compute_hh_trafo_kernel_complex_double<16><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 8:
        compute_hh_trafo_kernel_complex_double<8><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 4:
        compute_hh_trafo_kernel_complex_double<4><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 2:
        compute_hh_trafo_kernel_complex_double<2><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 1:
        compute_hh_trafo_kernel_complex_double<1><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    }

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("\n compute_hh_trafo CUDA kernel failed: %s \n",hipGetErrorString(err));
    }
}

template <unsigned int blk>
__global__ void compute_hh_trafo_kernel_complex_single(hipFloatComplex * __restrict__ q, const hipFloatComplex * __restrict__ hh, const hipFloatComplex * __restrict__ hh_tau, const int nb, const int ldq, const int ncols)
{
    __shared__ hipFloatComplex q_s[blk + 1];
    __shared__ hipFloatComplex dotp_s[blk];

    hipFloatComplex q_v2;

    int q_off, h_off, j;

    unsigned int tid = threadIdx.x;
    unsigned int bid = blockIdx.x;

    j = ncols;
    q_off = bid + (j + tid) * ldq;
    h_off = tid + j * nb;
    q_s[tid] = q[q_off];

    while (j >= 1)
    {
        if (tid == 0)
        {
            q_s[tid] = q[q_off];
        }

        q_v2 = q_s[tid];
        dotp_s[tid] = hipCmulf(q_v2, hipConjf(hh[h_off]));

        __syncthreads();

        reduce_complex<hipFloatComplex, blk>(dotp_s);

        __syncthreads();

        q_v2 = hipCsubf(q_v2, hipCmulf(hipCmulf(dotp_s[0], hh_tau[j]), hh[h_off]));
        q_s[tid + 1] = q_v2;

        if ((j == 1) || (tid == blockDim.x - 1))
        {
            q[q_off] = q_v2;
        }

        __syncthreads();

        q_off -= ldq;
        h_off -= nb;
        j -= 1;
    }
}

extern "C" void launch_compute_hh_trafo_c_kernel_complex_single(hipFloatComplex *q, const hipFloatComplex *hh, const hipFloatComplex *hh_tau, const int nev, const int nb, const int ldq, const int ncols)
{
    hipError_t err;

    switch (nb)
    {
    case 1024:
        compute_hh_trafo_kernel_complex_single<1024><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 512:
        compute_hh_trafo_kernel_complex_single<512><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 256:
        compute_hh_trafo_kernel_complex_single<256><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 128:
        compute_hh_trafo_kernel_complex_single<128><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 64:
        compute_hh_trafo_kernel_complex_single<64><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 32:
        compute_hh_trafo_kernel_complex_single<32><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 16:
        compute_hh_trafo_kernel_complex_single<16><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 8:
        compute_hh_trafo_kernel_complex_single<8><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 4:
        compute_hh_trafo_kernel_complex_single<4><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 2:
        compute_hh_trafo_kernel_complex_single<2><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    case 1:
        compute_hh_trafo_kernel_complex_single<1><<<nev, nb>>>(q, hh, hh_tau, nb, ldq, ncols);
        break;
    }

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("\n compute_hh_trafo CUDA kernel failed: %s \n",hipGetErrorString(err));
    }
}
